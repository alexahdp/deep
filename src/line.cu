#include "hip/hip_runtime.h"
#include <stdio.h>
#include <glew.h>
#include <GLFW/glfw3.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>


#include "line.cuh"
#include "lineShader.hpp"
#include "lib.hpp"

int LineStructSize = (int)sizeof(LineStruct);

__global__ void moveLines(LineStruct* line, LineFTStruct* ft, PointStruct* point, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i > n) return;
    
    line[i].pos.f = point[ ft[i].f ].pos;
    line[i].pos.t = point[ ft[i].t ].pos;
}

Line::Line(int _count) {
    this->count = _count;
    this->size = LineStructSize * _count;
    this->data = (LineStruct*)malloc(this->size);
    this->ft = (LineFTStruct*)malloc(sizeof(LineFTStruct) * _count);
    
    this->data[0].pos.f = {0, 0, 0};
    this->data[0].pos.t = {0.5, 0.5, 0};
    
    this->ft[0].f = 0;
    this->ft[0].t = 1;
    
    this->ftdptr = NULL;
    checkCudaErrors(hipMalloc((void **)&this->ftdptr, (int)sizeof(LineFTStruct) * _count));
    checkCudaErrors(hipMemcpy(this->ftdptr, this->ft, sizeof(LineFTStruct) * _count, hipMemcpyHostToDevice));
    
    // for (int i = 0; i < this->count; i++) {
    //     this->data[i].pos.f = {trand(), trand(), 0};
    //     this->data[i].pos.t = {trand(), trand(), 0};
    // }
    
    glGenBuffers(1, &this->VBO);
    
    this->dptr = NULL;
    this->lineShaderProgram = getLineShaderProgram();
    
    glBindBuffer(GL_ARRAY_BUFFER, this->VBO);
    glBufferData(GL_ARRAY_BUFFER, this->size, this->data, GL_DYNAMIC_DRAW);
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&this->cuda_vbo_resource, this->VBO, cudaGraphicsMapFlagsNone));
    glBindBuffer(GL_ARRAY_BUFFER, 0);
}

void Line::add(int f, int t) {
    this->count++;
    
    this->d2h();
    
    // а вот тут мне уже надо работать не с this->data, 
    // а с this->ft
    
    this->h2d();
}

void Line::bindVBO() {
    checkCudaErrors(hipGraphicsMapResources(1, &this->cuda_vbo_resource, 0));
    
    size_t num_bytes;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&this->dptr, &num_bytes, this->cuda_vbo_resource));
}

void Line::unbindVBO() {
    checkCudaErrors(hipGraphicsUnmapResources(1, &this->cuda_vbo_resource, 0));
}

void Point::d2h() {
    this->bindVBO();
    
    checkCudaErrors(hipMemcpy((void *)this->data, this->dptr, this->size(), hipMemcpyDeviceToHost));
    this->unbindVBO();
}

void Point::h2d() {
    this->bindVBO();
    checkCudaErrors(hipMemcpy(this->dptr, (void *)this->data, this->size(), hipMemcpyHostToDevice));
    this->unbindVBO();
}

void Line::draw() {
    glEnable(GL_LINE_SMOOTH);
    glUseProgram(this->lineShaderProgram);
    glLineWidth(2.0);
    
    //glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, LineStructSize, (GLvoid *)0);
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 0, (GLvoid *)0);
    
    //glVertexPointer(3, GL_FLOAT, LineStructSize, 0);
    glVertexPointer(3, GL_FLOAT, 0, 0);
    
    //glInterleavedArrays(GL_V2F, 0, NULL);
    
    glDrawArrays(GL_LINES, 0, this->count * 2);
    
    glDisable(GL_LINE_SMOOTH);
}

void Line::tick(Point* p1) {
    int THREADS_PER_BLOCK = 1024;
    
    int threads = this->count % THREADS_PER_BLOCK;
    int blocks = (this->count + threads) / threads;
    
    moveLines<<<blocks, threads>>>(this->dptr, this->ftdptr, p1->dptr, this->count);
}