#include "hip/hip_runtime.h"
#include <stdio.h>
#include <glew.h>
#include <GLFW/glfw3.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop

#include "pointShader.hpp"
//#include "point.cuh"

struct Point {
    float3 pos;
};


__device__ float3 operator+(const float3 &a, const float3 &b) {
  return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);

}

__global__ void simple_vbo_kernel(Point *point) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    point[i].pos = point[i].pos + make_float3(0.001, 0.001, 0);
}


class Ppoint {
    public:
        int size;
        int count;
        GLuint pointShaderProgram;
        GLuint VBO;
        struct hipGraphicsResource *cuda_vbo_resource;
        Point* data;
        Ppoint(int _count);
        Point *dptr;
        
        void bindVBO();
        void unbindVBO();
        void draw();
        void tick();
};

//int Ppoint::pointSize = sizeof(Point);

Ppoint::Ppoint(int _count) {
    this->count = _count;
    this->size = sizeof(Point) * _count;
    this->data = (Point*)malloc(this->size);
    
    this->data[0].pos = {0.1, 0.1, 0};
    this->data[1].pos = {0, 0.1, 0};
    this->data[2].pos = {0.1, 0, 0};
    
    this->dptr = NULL;
    
    this->pointShaderProgram = getPointShaderProgram();
}

void Ppoint::bindVBO() {
    checkCudaErrors(hipGraphicsMapResources(1, &this->cuda_vbo_resource, 0));
    
    size_t num_bytes;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&this->dptr, &num_bytes, this->cuda_vbo_resource));
}

void Ppoint::unbindVBO() {
    checkCudaErrors(hipGraphicsUnmapResources(1, &this->cuda_vbo_resource, 0));
}

void Ppoint::draw() {
    glUseProgram(this->pointShaderProgram);
    glPointSize(10.0);
    glVertexPointer(3, GL_FLOAT, 0, NULL);
    //glVertexPointer(3, GL_FLOAT, 0, dptr);
    
    //glDrawArrays(GL_POINTS, 0, (pointSize / sizeof(float)) / 3);
    glDrawArrays(GL_POINTS, 0, 3);
}

void Ppoint::tick() {
    //int blocks = sizeof(&point) / pointSize;
    simple_vbo_kernel<<<1, 9>>>(this->dptr);
}