#include "hip/hip_runtime.h"
#include <stdio.h>
#include <glew.h>
#include <GLFW/glfw3.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

#include "point.cuh"
#include "pointShader.hpp"
#include "lib.hpp"


__global__ void simple_vbo_kernel(PointStruct *point, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i > n) return;
    point[i].pos = point[i].pos + point[i].vel;
}


Point::Point(int _count) {
    this->count = _count;
    this->size = sizeof(PointStruct) * _count;
    this->data = (PointStruct*)malloc(this->size);
    
    for (int i = 0; i < this->count; i++) {
        this->data[i].pos = {trand(), trand(), 0};
        this->data[i].vel = {trand() / 500.0f, trand() / 500.0f, 0};
    }
    
    // this->data[0].pos = {0.5, 0, 0};
    // this->data[0].vel = {0, 0.001, 0};
    
    // this->data[1].pos = {-0.75, 0, 0};
    // this->data[1].vel = {0, -0.001, 0};
    
    // this->data[2].pos = {0, 0.5, 0};
    // this->data[2].vel = {0.001, 0, 0};
    
    // this->data[3].pos = {0, -0.5, 0};
    // this->data[3].vel = {-0.001, 0, 0};
    
    this->dptr = NULL;
    
    this->pointShaderProgram = getPointShaderProgram();
}

void Point::bindVBO() {
    checkCudaErrors(hipGraphicsMapResources(1, &this->cuda_vbo_resource, 0));
    
    size_t num_bytes;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&this->dptr, &num_bytes, this->cuda_vbo_resource));
}

void Point::unbindVBO() {
    checkCudaErrors(hipGraphicsUnmapResources(1, &this->cuda_vbo_resource, 0));
}

void Point::draw() {
    glUseProgram(this->pointShaderProgram);
    glPointSize(10.0);
    
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, sizeof(PointStruct), (GLvoid *)0);
    glDrawArrays(GL_POINTS, 0, this->count);
}

void Point::tick() {
    int THREADS_PER_BLOCK = 1024;
    
    int threads = this->count % THREADS_PER_BLOCK;
    int blocks = (this->count + threads) / threads;
    
    simple_vbo_kernel<<<blocks, threads>>>(this->dptr, this->count);
}
