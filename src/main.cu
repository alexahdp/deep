#include "hip/hip_runtime.h"
//
//  Полезные ссылки
//    https://solarianprogrammer.com/2013/05/13/opengl-101-drawing-primitives/
//    http://vbomesh.blogspot.ru/2012/02/vbo-opengl.html
//    https://www.khronos.org/opengl/wiki/VBO_-_just_examples
//    http://pmg.org.ru/nehe/
//    http://www.codenet.ru/progr/opengl/


// #define GLEW_STATIC
#include <stdio.h>
#include <math.h>
#include <glew.h>
#include <GLFW/glfw3.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop

#include <ctime>

//#include "lineShader.hpp"
//#include "pointShader.hpp"
#include "point.cuh"
#include "line.cuh"
#include "lib.hpp"

GLuint VAO;
GLuint VBOS[2];

const GLuint WIDTH = 800, HEIGHT = 600;
GLFWwindow* window;

Point* p1;
Line* l1;

void key_callback(GLFWwindow* window, int key, int scancode, int action, int mode) {
    
    if (key == GLFW_KEY_ESCAPE && action == GLFW_PRESS) {
        glfwSetWindowShouldClose(window, GL_TRUE);
    }
    
    std::cout << key << std::endl;
    std::cout << "===" << std::endl;
    
    if (action == GLFW_PRESS) {
        float3 pos = {trand(), trand(), 0};
        float3 vel = {trand(), trand(), 0};
        
        p1->add(pos, vel);
    }
}


int init() {
    std::cout << "Starting GLFW context, OpenGL 4.5" << std::endl;
    
    glfwInit();
    
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 4);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 5);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
    glfwWindowHint(GLFW_RESIZABLE, GL_FALSE);
    
    // Create a GLFWwindow object that we can use for GLFW's functions
    window = glfwCreateWindow(WIDTH, HEIGHT, "LearnOpenGL", nullptr, nullptr);
    if (window == nullptr) {
        std::cout << "Failed to create GLFW window" << std::endl;
        glfwTerminate();
        return -1;
    }
    
    glfwMakeContextCurrent(window);
    glfwSetKeyCallback(window, key_callback);
    
    glewExperimental = GL_TRUE;
    
    GLenum glewinit = glewInit();
    if (glewinit != GLEW_OK) {
        std::cout << "Failed to initialize GLEW" << glewGetErrorString(glewinit) << std::endl;
        return -1;
    }
    
    int width, height;
    glfwGetFramebufferSize(window, &width, &height);
    glViewport(0, 0, width, height);
    
    return 0;
}


void loop(Point* p1, Line* l1) {
    glfwPollEvents();
    glClearColor(0, 0, 0, 1.0f);
    glClear(GL_COLOR_BUFFER_BIT);
    
    //glEnable(GL_DEPTH_TEST);
    glEnableClientState(GL_VERTEX_ARRAY);
    
    
    glBindBuffer(GL_ARRAY_BUFFER, p1->VBO);
    //glVertexPointer(3, GL_FLOAT, 0, (GLvoid *)0);
    glEnableVertexAttribArray(0);
    p1->draw();
    glDisableVertexAttribArray(0);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    
    
    glBindBuffer(GL_ARRAY_BUFFER, l1->VBO);
    glEnableVertexAttribArray(0);
    l1->draw();
    glDisableVertexAttribArray(0);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    
    glDisableClientState(GL_VERTEX_ARRAY);
    
    glfwSwapBuffers(window);
    
    p1->bindVBO();
    p1->tick();
    p1->unbindVBO();
    
    l1->bindVBO();
    l1->tick();
    l1->unbindVBO();
}


int main() {
    
    std::srand(unsigned(std::time(0)));
    
    if (init() != 0) return 1;
    
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(0, WIDTH, 0, HEIGHT, 0, 1);
    glMatrixMode( GL_MODELVIEW );
    glLoadIdentity();
    
    glGenVertexArrays(1, &VAO);
    glBindVertexArray(VAO);
    
    p1 = new Point(2);
    l1 = new Line(1);
    
    cudaGLSetGLDevice(gpuGetMaxGflopsDeviceId());
    
    glGenBuffers(2, VBOS);
    p1->VBO = VBOS[0];
    l1->VBO = VBOS[1];
    
    glBindBuffer(GL_ARRAY_BUFFER, p1->VBO);
    glBufferData(GL_ARRAY_BUFFER, p1->SIZE, p1->data, GL_DYNAMIC_DRAW);
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&p1->cuda_vbo_resource, p1->VBO, cudaGraphicsMapFlagsNone));
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    
    glBindBuffer(GL_ARRAY_BUFFER, l1->VBO);
    glBufferData(GL_ARRAY_BUFFER, l1->size, l1->data, GL_DYNAMIC_DRAW);
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&l1->cuda_vbo_resource, l1->VBO, cudaGraphicsMapFlagsNone));
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    
    while (!glfwWindowShouldClose(window)) loop(p1, l1);
    
    checkCudaErrors(hipGraphicsUnregisterResource(p1->cuda_vbo_resource));
    checkCudaErrors(hipGraphicsUnregisterResource(l1->cuda_vbo_resource));
    
    glDeleteVertexArrays(1, &VAO);
    glDeleteBuffers(1, &p1->VBO);
    glDeleteBuffers(1, &l1->VBO);
    
    hipFree(p1->dptr);
    hipFree(l1->dptr);
    
    hipDeviceReset();
    glfwTerminate();
    
    return 0;
}