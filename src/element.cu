#include <stdio.h>
#include <glew.h>
#include <GLFW/glfw3.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

#include "element.cuh"

Element::Element() {
}

void Element::bindVBO() {
    checkCudaErrors(hipGraphicsMapResources(1, &this->cuda_vbo_resource, 0));
    
    size_t num_bytes;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&this->dptr, &num_bytes, this->cuda_vbo_resource));
}

void Element::unbindVBO() {
    checkCudaErrors(hipGraphicsUnmapResources(1, &this->cuda_vbo_resource, 0));
}

// void Element::d2h() {
//     this->bindVBO();
    
//     checkCudaErrors(hipMemcpy((void *)this->data, this->dptr, this->size(), hipMemcpyDeviceToHost));
//     this->unbindVBO();
// }

// void Element::h2d() {
//     this->bindVBO();
//     checkCudaErrors(hipMemcpy(this->dptr, (void *)this->data, this->size(), hipMemcpyHostToDevice));
//     this->unbindVBO();
// }